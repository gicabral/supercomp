#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/functional.h>
#include <thrust/transform.h>
#include <iostream>
#include <math.h> 
#include <thrust/iterator/constant_iterator.h>

int main() {
    thrust::device_vector<double> AAPL;
    thrust::device_vector<double> MSFT;
    
    double stocks_AAPL, stocks_MSFT, mean, var;

    while(std::cin){
        std::cin >> stocks_AAPL >> stocks_MSFT;
        AAPL.push_back(stocks_AAPL);
        MSFT.push_back(stocks_MSFT);
    }

    int N = AAPL.size();
    thrust::device_vector<double> MEAN_DIF(N);

    thrust::transform(AAPL.begin(), AAPL.end(), MSFT.begin(), MEAN_DIF.begin(), thrust::minus<double>());
    double val = thrust::reduce(MEAN_DIF.begin(), MEAN_DIF.end());
    mean = val/2517;
    std::cout << "Média: " << fabs(mean) << "\n";

    // thrust::device_vector<double> MV(N, mean);
    thrust::device_vector<double> MV_DIF(N);

    thrust::transform(MEAN_DIF.begin(), MEAN_DIF.end(), thrust::constant_iterator<double>(mean), MV_DIF.begin(), thrust::minus<double>());
    // thrust::transform(MEAN_DIF.begin(), MEAN_DIF.end(), MV.begin(), MV_DIF.begin(), thrust::minus<double>());
    thrust::transform(MV_DIF.begin(), MV_DIF.end(), MV_DIF.begin(), MV_DIF.begin(), thrust::multiplies<double>());
    double val2 = thrust::reduce(MV_DIF.begin(), MV_DIF.end());
    var = val2/N;
    std::cout << "Variância: " << var << "\n";

}
